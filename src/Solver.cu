#include "hip/hip_runtime.h"
#include "Solver.h"
#include <hipsparse.h>
#include <hipsolver.h>

#if 0
#define CUDA_CHECK(err) do { \
    hipError_t _e = (err); \
    if (_e != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(_e)); \
        std::abort(); \
    } \
} while(0)

#define CUSPARSE_CHECK(stat) do { \
    hipsparseStatus_t _s = (stat); \
    if (_s != HIPSPARSE_STATUS_SUCCESS) { \
        fprintf(stderr, "cuSPARSE error %s:%d: %d\n", __FILE__, __LINE__, int(_s)); \
        std::abort(); \
    } \
} while(0)
#endif

#define CUSOLVER_CHECK(stat) do { \
    hipsolverStatus_t _s = (stat); \
    if (_s != HIPSOLVER_STATUS_SUCCESS) { \
        fprintf(stderr, "cuSOLVER error %s:%d: %d\n", __FILE__, __LINE__, int(_s)); \
        std::abort(); \
    } \
} while(0)

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        std::abort();                                                          \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        std::abort();                                                          \
    }                                                                          \
}



Solver::Solver(Assembler &assembler): m_assembler(assembler)
{
    m_solVector.setZero(assembler.numDofs());
}

Solver::~Solver()
{
}

bool Solver::solveSingleIteration()
{
#if 0
    const DeviceMatrix<double>& Adev = m_assembler.matrix();
    const int m = Adev.rows();
    const int n = Adev.cols();
    assert(m == n && "This example expects a square system.");

    void* dA = const_cast<void*>(static_cast<const void*>(Adev.data()));

    double* d_csr_offsets = nullptr;
    CUDA_CHECK( hipMalloc((void**) &d_csr_offsets,
                           (m + 1) * sizeof(double)) );

    hipsparseHandle_t cusparseH = nullptr;
    CUSPARSE_CHECK(hipsparseCreate(&cusparseH));
    
    hipsparseDnMatDescr_t dnA = nullptr;
    const int64_t rows = m, cols = n, ld = n;
    CUSPARSE_CHECK(hipsparseCreateDnMat(&dnA, 
                                       rows, cols, ld,
                                       dA,
                                       HIP_R_64F,
                                       HIPSPARSE_ORDER_ROW));

    hipsparseSpMatDescr_t spA = nullptr;
    int64_t csrRows = rows, csrCols = cols, csrNNZ = 0;
    CUSPARSE_CHECK(hipsparseCreateCsr(&spA, csrRows, csrCols, csrNNZ,
                                     d_csr_offsets, nullptr, nullptr,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_64F));

    size_t bufferSize = 0;
    void*  dBuffer    = nullptr;

    CUSPARSE_CHECK(hipsparseDenseToSparse_bufferSize(
        cusparseH, dnA, spA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize));

    CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    CUSPARSE_CHECK(hipsparseDenseToSparse_analysis(
        cusparseH, dnA, spA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer));

    CUDA_CHECK(hipFree(dBuffer));
    CUDA_CHECK(hipFree(d_csr_offsets));
    CUSPARSE_CHECK(hipsparseDestroySpMat(spA));
    CUSPARSE_CHECK(hipsparseDestroyDnMat(dnA));
    CUSPARSE_CHECK(hipsparseDestroy(cusparseH));
#endif
    m_assembler.assemble(m_solVector);

    const DeviceMatrix<double>& Adev = m_assembler.matrix();
    const DeviceVector<double>& bdev = m_assembler.rhs();
    int num_rows = Adev.rows();
    int num_cols = Adev.cols();
    int ld       = num_cols;
    DeviceVector<double> solutionVector(num_rows);

    //std::cout << "m_solVector:\n";
    //m_solVector.print();
    //std::cout << "bdev:\n";
    //bdev.print();
    //std::cout << "Adev:\n";
    //Adev.print();
    //--------------------------------------------------------------------------
    // Device memory management
    int   *d_csr_offsets, *d_csr_columns;
    double *d_csr_values;
    CHECK_CUDA( hipMalloc((void**) &d_csr_offsets, (num_rows + 1) * sizeof(int)) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, Adev.data(),
                                        HIP_R_64F, HIPSPARSE_ORDER_ROW) )
    // Create sparse matrix B in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, num_rows, num_cols, 0,
                                      d_csr_offsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                        handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) )
    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &d_csr_columns, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &d_csr_values,  nnz * sizeof(float)) )
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns,
                                           d_csr_values) )
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(handle, matA, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )

    hipsolverSpHandle_t solverH = nullptr;
    CUSOLVER_CHECK(hipsolverSpCreate(&solverH));
    hipsparseMatDescr_t descrA_legacy = nullptr;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA_legacy))
    CHECK_CUSPARSE(hipsparseSetMatType(descrA_legacy, HIPSPARSE_MATRIX_TYPE_GENERAL))
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA_legacy, HIPSPARSE_INDEX_BASE_ZERO))
    double tol = 1e-12;
    int    reorder = 1;
    int singularity = -1;
    CUSOLVER_CHECK(hipsolverSpDcsrlsvchol(solverH,
                                        num_rows,
                                        nnz,
                                        descrA_legacy,
                                        d_csr_values,
                                        d_csr_offsets,
                                        d_csr_columns,
                                        bdev.data(),
                                        tol,
                                        reorder,
                                        solutionVector.data(),
                                        &singularity));
    if (singularity >= 0)
        printf("WARNING: The matrix is singular at row %d under tol %E\n", singularity, tol);

    double* host_sol = new double[num_rows];
    double* host_residual = new double[num_rows];
    CHECK_CUDA( hipMemcpy(host_sol, solutionVector.data(), sizeof(double)*num_rows, hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(host_residual, bdev.data(), sizeof(double)*num_rows, hipMemcpyDeviceToHost) )
    Eigen::Map<Eigen::VectorXd> solvec(host_sol, num_rows);
    Eigen::Map<Eigen::VectorXd> resvec(host_residual, num_rows);

    m_updateNorm = solvec.norm();
    m_residualNorm = resvec.norm();
    m_solVector = m_solVector + solutionVector;

    if (m_numIterations == 0)
    {
        m_initResidualNorm = m_residualNorm;
        m_initUpdateNorm = m_updateNorm;
    }

    m_numIterations++;

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CUSOLVER_CHECK(hipsolverSpDestroy(solverH));
    CHECK_CUSPARSE( hipsparseDestroyMatDescr(descrA_legacy) )
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(d_csr_offsets) )
    CHECK_CUDA( hipFree(d_csr_columns) )
    CHECK_CUDA( hipFree(d_csr_values) )
    delete[] host_sol;
    delete[] host_residual;


    return true;
}

void Solver::solve()
{
    double absTol = 1e-10;
    double relTol = 1e-9;
    int maxIterations = 50;
    m_numIterations = 0;
    m_status = working;
    while (m_status == working)
    {
        if(!solveSingleIteration())
        {
            m_status = bad_solution;
            break;
        }
        std::cout << status() << std::endl;
        if (m_residualNorm < absTol || 
            m_updateNorm < absTol || 
            m_residualNorm/m_initResidualNorm < relTol ||
            m_updateNorm/m_initUpdateNorm < relTol)
            m_status = converged;
        else if (m_numIterations >= maxIterations)
            m_status = interrupted;
    }

    std::cout << status() << std::endl;
}

std::string Solver::status()
{
    std::string statusString;
if (m_status == converged)
    statusString = "Iterative solver converged after " +
             std::to_string(m_numIterations) + " iteration(s).";
else if (m_status == interrupted)
    statusString = "Iterative solver was interrupted after " +
            std::to_string(m_numIterations) + " iteration(s).";
else if (m_status == bad_solution)
    statusString = "Iterative solver was interrupted after " +
            std::to_string(m_numIterations) + " iteration(s) due to an invalid solution";
else if (m_status == working)
    statusString = "It: " + std::to_string(m_numIterations) +
             ", updAbs: " + std::to_string(m_updateNorm) +
             ", updRel: " + std::to_string(m_updateNorm/m_initUpdateNorm) +
             ", resAbs: " + std::to_string(m_residualNorm) +
             ", resRel: " + std::to_string(m_residualNorm/m_initResidualNorm);
return statusString;
}
